#include <hip/hip_runtime_api.h>
#include "example.h"

void ExampleMember::Gpu(hipStream_t stream, Array<const float>& arg1,
                        Array<const int>& arg2, Array<float>& out) {
  hipMemcpy(out.ptr, arg1.ptr, sizeof(float) * arg1.spec->Size(), hipMemcpyDeviceToDevice);
}

void Example::Gpu(hipStream_t stream, Array<const float>& arg1,
                  Array<const int>& arg2, Array<float>& out) {
  hipMemcpy(out.ptr, arg1.ptr, sizeof(float) * arg1.spec->Size(), hipMemcpyDeviceToDevice);
}
